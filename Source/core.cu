/*
	Copyright (c) 2011, T. Kroes <t.kroes@tudelft.nl>
	All rights reserved.

	Redistribution and use in source and binary forms, with or without modification, are permitted provided that the following conditions are met:

	- Redistributions of source code must retain the above copyright notice, this list of conditions and the following disclaimer.
	- Redistributions in binary form must reproduce the above copyright notice, this list of conditions and the following disclaimer in the documentation and/or other materials provided with the distribution.
	- Neither the name of the TU Delft nor the names of its contributors may be used to endorse or promote products derived from this software without specific prior written permission.
	
	THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

#define __CUDA_ARCH__ 200

texture<unsigned short, 3, hipReadModeNormalizedFloat> VolumeTexture;

#include "tracer.h"
#include "volume.h"
#include "light.h"
#include "object.h"
#include "clippingobject.h"
#include "texture.h"
#include "bitmap.h"

DEVICE ExposureRender::Tracer*				gpTracer				= NULL;
DEVICE ExposureRender::Volume* 				gpVolumes				= NULL;
DEVICE ExposureRender::Light*				gpLights				= NULL;
DEVICE ExposureRender::Object*				gpObjects				= NULL;
DEVICE ExposureRender::ClippingObject*		gpClippingObjects		= NULL;
DEVICE ExposureRender::Texture*				gpTextures				= NULL;
DEVICE ExposureRender::Bitmap*				gpBitmaps				= NULL;

#include "list.cuh"

ExposureRender::Cuda::List<ExposureRender::Tracer, ExposureRender::ErTracer>					gTracers("gpTracer");
ExposureRender::Cuda::List<ExposureRender::Volume, ExposureRender::ErVolume>					gVolumes("gpVolumes");
ExposureRender::Cuda::List<ExposureRender::Light, ExposureRender::ErLight>						gLights("gpLights");
ExposureRender::Cuda::List<ExposureRender::Object, ExposureRender::ErObject>					gObjects("gpObjects");
ExposureRender::Cuda::List<ExposureRender::ClippingObject, ExposureRender::ErClippingObject>	gClippingObjects("gpClippingObjects");
ExposureRender::Cuda::List<ExposureRender::Texture, ExposureRender::ErTexture>					gTextures("gpTextures");
ExposureRender::Cuda::List<ExposureRender::Bitmap, ExposureRender::ErBitmap>					gBitmaps("gpBitmaps");

#include "autofocus.cuh"
#include "singlescattering.cuh"
#include "filterframeestimate.cuh"
#include "estimate.cuh"
#include "toneMap.cuh"

namespace ExposureRender
{

EXPOSURE_RENDER_DLL void BindTracer(const ErTracer& Tracer, const bool& Bind /*= true*/)
{
	DebugLog("%s, Bind = %s", __FUNCTION__, Bind ? "true" : "false");
	
	if (Bind)
		gTracers.Bind(Tracer);
	else
		gTracers.Unbind(Tracer);
}

EXPOSURE_RENDER_DLL void BindVolume(const ErVolume& Volume, const bool& Bind /*= true*/)
{
	DebugLog("%s, Bind = %s", __FUNCTION__, Bind ? "true" : "false");

	if (Bind)
		gVolumes.Bind(Volume);
	else
		gVolumes.Unbind(Volume);
}

EXPOSURE_RENDER_DLL void BindLight(const ErLight& Light, const bool& Bind /*= true*/)
{
	DebugLog("%s, Bind = %s", __FUNCTION__, Bind ? "true" : "false");
	
	if (Bind)
		gLights.Bind(Light);
	else
		gLights.Unbind(Light);
}

EXPOSURE_RENDER_DLL void BindObject(const ErObject& Object, const bool& Bind /*= true*/)
{
	DebugLog("%s, Bind = %s", __FUNCTION__, Bind ? "true" : "false");
	
	if (Bind)
		gObjects.Bind(Object);
	else
		gObjects.Unbind(Object);
}

EXPOSURE_RENDER_DLL void BindClippingObject(const ErClippingObject& ClippingObject, const bool& Bind /*= true*/)
{
	DebugLog("%s, Bind = %s", __FUNCTION__, Bind ? "true" : "false");
	
	if (Bind)
		gClippingObjects.Bind(ClippingObject);
	else
		gClippingObjects.Unbind(ClippingObject);
}

EXPOSURE_RENDER_DLL void BindTexture(const ErTexture& Texture, const bool& Bind /*= true*/)
{
	DebugLog("%s, Bind = %s", __FUNCTION__, Bind ? "true" : "false");
	
	if (Bind)
		gTextures.Bind(Texture);
	else
		gTextures.Unbind(Texture);
}

EXPOSURE_RENDER_DLL void BindBitmap(const ErBitmap& Bitmap, const bool& Bind /*= true*/)
{
	DebugLog("%s, Bind = %s", __FUNCTION__, Bind ? "true" : "false");
	
	if (Bind)
		gBitmaps.Bind(Bitmap);
	else
		gBitmaps.Unbind(Bitmap);
}

EXPOSURE_RENDER_DLL void Render(int TracerID)
{
	if (gTracers[TracerID].NoEstimates == 0)
	{
		if (gTracers[TracerID].Camera.FocusMode == Enums::AutoFocus)
		{
			float AutoFocusDistance = -1.0f;

			ComputeAutoFocusDistance(gTracers[TracerID].Camera.FocusUV[0] * (float)gTracers[TracerID].FrameBuffer.Resolution[0], gTracers[TracerID].Camera.FocusUV[1] * (float)gTracers[TracerID].FrameBuffer.Resolution[1], AutoFocusDistance);

			if (AutoFocusDistance >= 0.0f)
				gTracers[TracerID].Camera.FocalDistance = AutoFocusDistance;
		}
	}

	gTracers.Synchronize(TracerID);

	gVolumes[gTracers[TracerID].VolumeID].Voxels.Bind(VolumeTexture);

	SingleScattering(gTracers[TracerID]);
	FilterFrameEstimate(gTracers[TracerID]);
	ComputeEstimate(gTracers[TracerID]);
	ToneMap(gTracers[TracerID]);

	gTracers[TracerID].NoEstimates++;
}

EXPOSURE_RENDER_DLL void GetRunningEstimate(int TracerID, ColorRGBAuc* pData)
{
	FrameBuffer& FB = gTracers[TracerID].FrameBuffer;

	Cuda::MemCopyDeviceToHost(FB.DisplayEstimate.GetData(), (ColorRGBAuc*)pData, FB.DisplayEstimate.GetNoElements());
}

}
