/*
	Copyright (c) 2011, T. Kroes <t.kroes@tudelft.nl>
	All rights reserved.

	Redistribution and use in source and binary forms, with or without modification, are permitted provided that the following conditions are met:

	- Redistributions of source code must retain the above copyright notice, this list of conditions and the following disclaimer.
	- Redistributions in binary form must reproduce the above copyright notice, this list of conditions and the following disclaimer in the documentation and/or other materials provided with the distribution.
	- Neither the name of the TU Delft nor the names of its contributors may be used to endorse or promote products derived from this software without specific prior written permission.
	
	THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

//#define __CUDA_ARCH__ 200

#include <map>

using namespace std;

texture<unsigned short, 3, hipReadModeNormalizedFloat> TexVolume0;
texture<unsigned short, 3, hipReadModeNormalizedFloat> TexVolume1;
texture<unsigned short, 3, hipReadModeNormalizedFloat> TexVolume2;
texture<unsigned short, 3, hipReadModeNormalizedFloat> TexVolume3;
texture<unsigned short, 3, hipReadModeNormalizedFloat> TexVolume4;

map<int, int> gTracersHashMap;
map<int, int> gVolumesHashMap;
map<int, int> gLightsHashMap;
map<int, int> gObjectsHashMap;
map<int, int> gClippingObjectsHashMap;
map<int, int> gTexturesHashMap;
map<int, int> gBitmapsHashMap;

#include "tracer.h"
#include "volume.h"
#include "light.h"
#include "object.h"
#include "clippingobject.h"
#include "texture.h"
#include "bitmap.h"

DEVICE ExposureRender::Tracer*			gpTracer			= NULL;
DEVICE ExposureRender::Volume* 			gpVolumes			= NULL;
DEVICE ExposureRender::Light*			gpLights			= NULL;
DEVICE ExposureRender::Object*			gpObjects			= NULL;
DEVICE ExposureRender::ClippingObject*	gpClippingObjects	= NULL;
DEVICE ExposureRender::Texture*			gpTextures			= NULL;
DEVICE ExposureRender::Bitmap*			gpBitmaps			= NULL;

#include "list.cuh"

ExposureRender::Cuda::List<ExposureRender::Tracer, ExposureRender::ErTracer>					gTracers("gpTracer");
ExposureRender::Cuda::List<ExposureRender::Volume, ExposureRender::ErVolume>					gVolumes("gpVolumes");
ExposureRender::Cuda::List<ExposureRender::Light, ExposureRender::ErLight>						gLights("gpLights");
ExposureRender::Cuda::List<ExposureRender::Object, ExposureRender::ErObject>					gObjects("gpObjects");
ExposureRender::Cuda::List<ExposureRender::ClippingObject, ExposureRender::ErClippingObject>	gClippingObjects("gpClippingObjects");
ExposureRender::Cuda::List<ExposureRender::Texture, ExposureRender::ErTexture>					gTextures("gpTextures");
ExposureRender::Cuda::List<ExposureRender::Bitmap, ExposureRender::ErBitmap>					gBitmaps("gpBitmaps");

#include "autofocus.cuh"
#include "singlescattering.cuh"
#include "filtering.cuh"
#include "estimate.cuh"
#include "toneMap.cuh"

namespace ExposureRender
{

EXPOSURE_RENDER_DLL void BindTracer(const ErTracer& Tracer, const bool& Bind /*= true*/)
{
	if (Bind)
		gTracers.Bind(Tracer);
	else
		gTracers.Unbind(Tracer);

	gTracersHashMap = gTracers.HashMap;
}

EXPOSURE_RENDER_DLL void BindVolume(const ErVolume& Volume, const bool& Bind /*= true*/)
{
	if (Bind)
	{
		ExposureRender::Volume* DeviceVolume = gVolumes.Bind(Volume);

		if (DeviceVolume)
		{
			DeviceVolume->Voxels.Bind(TexVolume0);
		}
	}
	else
		gVolumes.Unbind(Volume);

	gVolumesHashMap = gVolumes.HashMap;
}

EXPOSURE_RENDER_DLL void BindLight(const ErLight& Light, const bool& Bind /*= true*/)
{
	if (Bind)
		gLights.Bind(Light);
	else
		gLights.Unbind(Light);

	gLightsHashMap = gLights.HashMap;
}

EXPOSURE_RENDER_DLL void BindObject(const ErObject& Object, const bool& Bind /*= true*/)
{
	if (Bind)
		gObjects.Bind(Object);
	else
		gObjects.Unbind(Object);

	gObjectsHashMap = gObjects.HashMap;
}

EXPOSURE_RENDER_DLL void BindClippingObject(const ErClippingObject& ClippingObject, const bool& Bind /*= true*/)
{
	if (Bind)
		gClippingObjects.Bind(ClippingObject);
	else
		gClippingObjects.Unbind(ClippingObject);

	gClippingObjectsHashMap = gClippingObjects.HashMap;
}

EXPOSURE_RENDER_DLL void BindTexture(const ErTexture& Texture, const bool& Bind /*= true*/)
{
	if (Bind)
		gTextures.Bind(Texture);
	else
		gTextures.Unbind(Texture);

	gTexturesHashMap = gTextures.HashMap;
}

EXPOSURE_RENDER_DLL void BindBitmap(const ErBitmap& Bitmap, const bool& Bind /*= true*/)
{
	if (Bind)
		gBitmaps.Bind(Bitmap);
	else
		gBitmaps.Unbind(Bitmap);

	gBitmapsHashMap = gBitmaps.HashMap;
}

EXPOSURE_RENDER_DLL void Render(int TracerID)
{
	if (gTracers[TracerID].NoEstimates == 0)
	{
		if (gTracers[TracerID].Camera.FocusMode == Enums::AutoFocus)
		{
			float AutoFocusDistance = -1.0f;

			const Vec2i FilmUV((int)(gTracers[TracerID].Camera.FocusUV[0] * (float)gTracers[TracerID].FrameBuffer.Resolution[0]), (int)(gTracers[TracerID].Camera.FocusUV[1] * (float)gTracers[TracerID].FrameBuffer.Resolution[1]));
			ComputeAutoFocusDistance(FilmUV, AutoFocusDistance);

			if (AutoFocusDistance >= 0.0f)
				gTracers[TracerID].Camera.FocalDistance = AutoFocusDistance;
		}
	}

	gTracers.Synchronize(TracerID);

	if (gTracers[TracerID].VolumeIDs[0] >= 0)
		gVolumes[gTracers[TracerID].VolumeIDs[0]].Voxels.Bind(TexVolume0);

	if (gTracers[TracerID].VolumeIDs[1] >= 0)
		gVolumes[gTracers[TracerID].VolumeIDs[1]].Voxels.Bind(TexVolume1);

	if (gTracers[TracerID].VolumeIDs[2] >= 0)
		gVolumes[gTracers[TracerID].VolumeIDs[2]].Voxels.Bind(TexVolume2);

	if (gTracers[TracerID].VolumeIDs[3] >= 0)
		gVolumes[gTracers[TracerID].VolumeIDs[3]].Voxels.Bind(TexVolume3);

	SingleScattering(gTracers[TracerID]);
	ComputeEstimate(gTracers[TracerID]);
	ToneMap(gTracers[TracerID]);
	GaussianFilterRunningEstimate(gTracers[TracerID]);
//	BilateralFilterRunningEstimate(gTracers[TracerID]);

	gTracers[TracerID].NoEstimates++;
}

EXPOSURE_RENDER_DLL void GetRunningEstimate(int TracerID, ColorRGBAuc* pData)
{
	FrameBuffer& FB = gTracers[TracerID].FrameBuffer;

	Cuda::MemCopyDeviceToHost(FB.DisplayEstimate.GetData(), (ColorRGBAuc*)pData, FB.DisplayEstimate.GetNoElements());
}

}
