#include "hip/hip_runtime.h"
/*
	Copyright (c) 2011, T. Kroes <t.kroes@tudelft.nl>
	All rights reserved.

	Redistribution and use in source and binary forms, with or without modification, are permitted provided that the following conditions are met:

	- Redistributions of source code must retain the above copyright notice, this list of conditions and the following disclaimer.
	- Redistributions in binary form must reproduce the above copyright notice, this list of conditions and the following disclaimer in the documentation and/or other materials provided with the distribution.
	- Neither the name of the TU Delft nor the names of its contributors may be used to endorse or promote products derived from this software without specific prior written permission.
	
	THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

#include <map>

using namespace std;

texture<unsigned short, 3, hipReadModeNormalizedFloat> TexVolume0;

#include "color.h"

texture<float, 1, hipReadModeElementType>			Opacity1D;
texture<float4, 1, hipReadModeElementType> 		Diffuse1D;
texture<float4, 1, hipReadModeElementType> 		Specular1D;
texture<float, 1, hipReadModeElementType> 			Glossiness1D;
texture<float, 1, hipReadModeElementType> 			IndexOfReflection1D;
texture<float4, 1, hipReadModeElementType> 		Emission1D;

map<int, int> gTracersHashMap;
map<int, int> gVolumesHashMap;
map<int, int> gObjectsHashMap;
map<int, int> gClippingObjectsHashMap;
map<int, int> gTexturesHashMap;
map<int, int> gBitmapsHashMap;

CONSTANT_DEVICE float gDensityScale			= 0.0f;
CONSTANT_DEVICE float gStepFactorPrimary	= 0.0f;
CONSTANT_DEVICE float gStepFactorShadow		= 0.0f;

#include "statistics.h"
#include "tracer.h"
#include "volume.h"
#include "light.h"
#include "object.h"
#include "clippingobject.h"
#include "texture.h"
#include "bitmap.h"

DEVICE ExposureRender::Tracer*			gpTracer			= NULL;
DEVICE ExposureRender::Volume* 			gpVolumes			= NULL;
DEVICE ExposureRender::Object*			gpObjects			= NULL;
DEVICE ExposureRender::ClippingObject*	gpClippingObjects	= NULL;
DEVICE ExposureRender::Texture*			gpTextures			= NULL;
DEVICE ExposureRender::Bitmap*			gpBitmaps			= NULL;

#include "list.cuh"

ExposureRender::Cuda::List<ExposureRender::Tracer, ExposureRender::ErTracer>					gTracers("gpTracer");
ExposureRender::Cuda::List<ExposureRender::Volume, ExposureRender::ErVolume>					gVolumes("gpVolumes");
ExposureRender::Cuda::List<ExposureRender::Object, ExposureRender::ErObject>					gObjects("gpObjects");
ExposureRender::Cuda::List<ExposureRender::ClippingObject, ExposureRender::ErClippingObject>	gClippingObjects("gpClippingObjects");
ExposureRender::Cuda::List<ExposureRender::Texture, ExposureRender::ErTexture>					gTextures("gpTextures");
ExposureRender::Cuda::List<ExposureRender::Bitmap, ExposureRender::ErBitmap>					gBitmaps("gpBitmaps");

#include "autofocus.cuh"
#include "singlescattering.cuh"
#include "filtering.cuh"
#include "estimate.cuh"
#include "tonemap.cuh"
#include "composite.cuh"
#include "emptyspace.cuh"

namespace ExposureRender
{

EXPOSURE_RENDER_DLL void BindTracer(const ErTracer& Tracer, const bool& Bind /*= true*/)
{
	if (Bind)
		gTracers.Bind(Tracer);
	else
		gTracers.Unbind(Tracer);

	gTracersHashMap = gTracers.HashMap;
}

EXPOSURE_RENDER_DLL void BindVolume(const ErVolume& Volume, const bool& Bind /*= true*/)
{
	if (Bind)
		gVolumes.Bind(Volume);
	else
		gVolumes.Unbind(Volume);

	gVolumesHashMap = gVolumes.HashMap;
}

EXPOSURE_RENDER_DLL void BindObject(const ErObject& Object, const bool& Bind /*= true*/)
{
	if (Bind)
		gObjects.Bind(Object);
	else
		gObjects.Unbind(Object);

	gObjectsHashMap = gObjects.HashMap;
}

EXPOSURE_RENDER_DLL void BindClippingObject(const ErClippingObject& ClippingObject, const bool& Bind /*= true*/)
{
	if (Bind)
		gClippingObjects.Bind(ClippingObject);
	else
		gClippingObjects.Unbind(ClippingObject);

	gClippingObjectsHashMap = gClippingObjects.HashMap;
}

EXPOSURE_RENDER_DLL void BindTexture(const ErTexture& Texture, const bool& Bind /*= true*/)
{
	if (Bind)
		gTextures.Bind(Texture);
	else
		gTextures.Unbind(Texture);

	gTexturesHashMap = gTextures.HashMap;
}

EXPOSURE_RENDER_DLL void BindBitmap(const ErBitmap& Bitmap, const bool& Bind /*= true*/)
{
	if (Bind)
		gBitmaps.Bind(Bitmap);
	else
		gBitmaps.Unbind(Bitmap);

	gBitmapsHashMap = gBitmaps.HashMap;
}

EXPOSURE_RENDER_DLL void Render(int TracerID, Statistics& Statistics)
{
	hipEvent_t EventStart, EventStop;

	Cuda::HandleCudaError(hipEventCreate(&EventStart));
	Cuda::HandleCudaError(hipEventCreate(&EventStop));
	Cuda::HandleCudaError(hipEventRecord(EventStart, 0));

	Tracer& Tracer = gTracers[TracerID];

	const float DensityScale		= Tracer.VolumeProperty.DensityScale;
	const float StepFactorPrimary	= gVolumes[gVolumesHashMap[Tracer.VolumeIDs[0]]].MinStep * Tracer.VolumeProperty.StepFactorPrimary;
	const float StepFactorShadow	= gVolumes[gVolumesHashMap[Tracer.VolumeIDs[0]]].MinStep * Tracer.VolumeProperty.StepFactorShadow;
	
	Cuda::HostToConstantDevice(&DensityScale, "gDensityScale");
	Cuda::HostToConstantDevice(&StepFactorPrimary, "gStepFactorPrimary");
	Cuda::HostToConstantDevice(&StepFactorShadow, "gStepFactorShadow");

	if (Tracer.NoEstimates == 0)
	{
		if (Tracer.Camera.FocusMode == Enums::AutoFocus)
		{
			float AutoFocusDistance = -1.0f;

			const Vec2i FilmUV((int)(Tracer.Camera.FocusUV[0] * (float)Tracer.FrameBuffer.Resolution[0]), (int)(Tracer.Camera.FocusUV[1] * (float)Tracer.FrameBuffer.Resolution[1]));
			ComputeAutoFocusDistance(FilmUV, AutoFocusDistance, Statistics);

			if (AutoFocusDistance >= 0.0f)
				Tracer.Camera.FocalDistance = AutoFocusDistance;
		}
	}

	gTracers.Synchronize(TracerID);

	if (Tracer.VolumeIDs[0] >= 0)
		gVolumes[Tracer.VolumeIDs[0]].Voxels.Bind(TexVolume0);

	SingleScattering(Tracer, Statistics);
	GaussianFilterFrameEstimate(Tracer, Statistics);
	ComputeEstimate(Tracer, Statistics);
	ToneMap(Tracer, Statistics);
	GaussianFilterRunningEstimate(Tracer, Statistics);
	
	if (Tracer.NoiseReduction)
		BilateralFilterRunningEstimate(Tracer, Statistics);
	
	Composite(Tracer, Statistics);

	Tracer.NoEstimates++;

	Cuda::HandleCudaError(hipEventRecord(EventStop, 0));
	Cuda::HandleCudaError(hipEventSynchronize(EventStop));
																							
	float TimeDelta = 0.0f;
																							
	Cuda::HandleCudaError(hipEventElapsedTime(&TimeDelta, EventStart, EventStop), "hipEventElapsedTime");
	
	Statistics.FPS = 1000.0f / TimeDelta;
														
	Cuda::HandleCudaError(hipEventDestroy(EventStart));
	Cuda::HandleCudaError(hipEventDestroy(EventStop));										

}

EXPOSURE_RENDER_DLL void GetDisplayEstimate(int TracerID, ColorRGBAuc* pData)
{
	FrameBuffer& FB = gTracers[TracerID].FrameBuffer;

	Cuda::MemCopyDeviceToHost(FB.DisplayEstimate.GetData(), (ColorRGBAuc*)pData, FB.DisplayEstimate.GetNoElements());
}

}
